#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Matrix.hpp"

namespace YuiOkusora
{
	namespace Cuda
	{
		namespace Matrix
		{
			__global__ static void __cudaAddMatrix(double* a, double *b, size_t rows, size_t cols)
			{
				unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;

				if (i < rows * cols)
					a[i] = a[i] + b[i];
				if (i < rows * cols - 1)
					a[i + 1] = a[i + 1] + b[i + 1];
			}

			__global__ static void __cudaAddVal2Matrix(double* a, double* b, size_t rows, size_t cols)
			{
				unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;

				if (i < rows * cols)
					a[i] = a[i] + *b;
				if (i < rows * cols - 1)
					a[i + 1] = a[i + 1] + *b;
			}

			__global__ static void __cudaSubtractMatrix(double* a, double* b, size_t rows, size_t cols)
			{
				unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;

				if (i < rows * cols)
					a[i] = a[i] - b[i];
				if (i < rows * cols - 1)
					a[i + 1] = a[i + 1] - b[i + 1];
			}
			
			__global__ static void __cudaTranspose(double* a, double* b, size_t rows, size_t cols)
			{
				unsigned i = (threadIdx.x + blockDim.x * blockIdx.x); // rows
				unsigned j = (threadIdx.y + blockDim.y * blockIdx.y); // cols

				if (i < rows && j < cols) {
					unsigned idx = i * cols + j;
					unsigned ridx = j * rows + i;

					b[ridx] = a[idx];
				}

			}

			__global__ static void __cudaMultiplyVal2Matrix(double* a, double* b, size_t rows, size_t cols)
			{
				unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;

				if (i < rows * cols)
					a[i] = a[i] * *b;
				if (i < rows * cols - 1)
					a[i + 1] = a[i + 1] * *b;
			}

			__global__ static void __cudaMultiplyMat2Matrix(double* a, double* b, size_t rows, size_t cols)
			{
				unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;

				if (i < rows * cols)
					a[i] = a[i] * b[i];
				if (i < rows * cols - 1)
					a[i + 1] = a[i + 1] * b[i + 1];
			}

			__global__ static void __cudaRotateMatrix180(double* a, size_t rows, size_t cols)
			{
				unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;
				double tmp = 0;
				const size_t size = rows * cols;
				if (i < size / 2)
				{
					tmp = a[i];
					a[i] = a[size - i - 1];
					a[size - i - 1] = tmp;
				}
					
				if (i < (size / 2) - 1)
				{
					tmp = a[i + 1];
					a[i + 1] = a[size - i - 2];
					a[size - i - 2] = tmp;
				}
			}

			__global__ static void __cudaDotProductMatrix(double* a, double* b, double* c, size_t Arows, size_t Acols, size_t Bcols)
			{
				unsigned i = threadIdx.x + blockIdx.x * blockDim.x; // rows
				unsigned j = threadIdx.y + blockIdx.y * blockDim.y; // cols
				
				if (i >= Arows || j >= Bcols) return;

				for (unsigned k = 0; k < Acols; ++k)
				{
					c[i * Bcols + j] += a[i * Acols + k] * b[k * Bcols + j];
				}
			}
		}
	}
	namespace Math
	{
		namespace Mat
		{
			void fillMatrix(Matrix* a, const double& b)
			{

			}

			void addMatrix(Matrix *a, Matrix &b)
			{
				double* aPtr, *bPtr;
				hipMalloc(&aPtr, sizeof(double) * a->getCols() * a->getRows());
				hipMalloc(&bPtr, sizeof(double) * a->getCols() * a->getRows());

				hipMemcpy(aPtr, a->getFlatted(), sizeof(double) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
				hipMemcpy(bPtr, b.getFlatted(), sizeof(double) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);

				unsigned blockNum = 1, threadsPerBlock = 0;
				for (int i = 0; i <= 5; ++i) {
					threadsPerBlock = 32 << i;
					blockNum = ceil(double(a->getCols() * a->getRows()) / double(threadsPerBlock * 2));
					if (blockNum <= 208) break;
				}

				YuiOkusora::Cuda::Matrix::__cudaAddMatrix<<<blockNum, threadsPerBlock>>>(aPtr, bPtr, a->getRows(), a->getCols());
				
				hipDeviceSynchronize();
				
				hipMemcpy(a->getFlatted(), aPtr, sizeof(double) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
				hipFree(aPtr);
				hipFree(bPtr);
				
			}

			void addVal2Matrix(Matrix* a, const double &b)
			{
				double* aPtr, * bPtr;
				hipMalloc(&aPtr, sizeof(double) * a->getCols() * a->getRows());
				hipMalloc(&bPtr, sizeof(double));

				hipMemcpy(aPtr, a->getFlatted(), sizeof(double) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
				hipMemcpy(bPtr, &b, sizeof(double), hipMemcpyHostToDevice);

				unsigned blockNum = 1, threadsPerBlock = 0;
				for (int i = 0; i <= 5; ++i) {
					threadsPerBlock = 32 << i;
					blockNum = ceil(double(a->getCols() * a->getRows()) / double(threadsPerBlock * 2));
					if (blockNum <= 208) break;
				}

				YuiOkusora::Cuda::Matrix::__cudaAddVal2Matrix << <blockNum, threadsPerBlock >> > (aPtr, bPtr, a->getRows(), a->getCols());

				hipDeviceSynchronize();

				hipMemcpy(a->getFlatted(), aPtr, sizeof(double) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
				hipFree(aPtr);
				hipFree(bPtr);

			}

			void subtractMatrix(Matrix* a, Matrix& b)
			{
				double* aPtr, * bPtr;
				hipMalloc(&aPtr, sizeof(double) * a->getCols() * a->getRows());
				hipMalloc(&bPtr, sizeof(double) * a->getCols() * a->getRows());

				hipMemcpy(aPtr, a->getFlatted(), sizeof(double) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
				hipMemcpy(bPtr, b.getFlatted(), sizeof(double) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);

				unsigned blockNum = 1, threadsPerBlock = 0;
				for (int i = 0; i <= 5; ++i) {
					threadsPerBlock = 32 << i;
					blockNum = ceil(double(a->getCols() * a->getRows()) / double(threadsPerBlock * 2));
					if (blockNum <= 208) break;
				}

				YuiOkusora::Cuda::Matrix::__cudaSubtractMatrix << <blockNum, threadsPerBlock >> > (aPtr, bPtr, a->getRows(), a->getCols());

				hipDeviceSynchronize();

				hipMemcpy(a->getFlatted(), aPtr, sizeof(double) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
				hipFree(aPtr);
				hipFree(bPtr);

			}
		
			void transposeMatrix(Matrix* a)
			{
				double* aPtr, *bPtr;
				hipMalloc(&aPtr, sizeof(double) * a->getCols() * a->getRows());
				hipMalloc(&bPtr, sizeof(double) * a->getCols() * a->getRows());

				hipMemcpy(aPtr, a->getFlatted(), sizeof(double) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);

				dim3 blockNum(0,0,1), threadNum(0,0,1);
				for (int i = 1; i <= 32;++i)
				{
					threadNum.x = i;
					blockNum.x = ceil(double(a->getRows()) / double(i));
					if (blockNum.x <= 208) break;
				}
				
				for (int i = 1; i <= 32;++i)
				{
					threadNum.y = i;
					blockNum.y = ceil(double(a->getCols()) / double(i));
					if (blockNum.y <= 208) break;
				}

				YuiOkusora::Cuda::Matrix::__cudaTranspose<<<blockNum, threadNum>>>(aPtr, bPtr, a->getRows(), a->getCols());

				hipDeviceSynchronize();

				hipMemcpy(a->getFlatted(), bPtr, sizeof(double) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);

				hipFree(aPtr);
				hipFree(bPtr);

			}
		
			void multiplyVal2Matrix(Matrix* a, const double& b)
			{
				double* aPtr, * bPtr;
				hipMalloc(&aPtr, sizeof(double) * a->getCols() * a->getRows());
				hipMalloc(&bPtr, sizeof(double));

				hipMemcpy(aPtr, a->getFlatted(), sizeof(double) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
				hipMemcpy(bPtr, &b, sizeof(double), hipMemcpyHostToDevice);

				unsigned blockNum = 1, threadsPerBlock = 0;
				for (int i = 0; i <= 5; ++i) {
					threadsPerBlock = 32 << i;
					blockNum = ceil(double(a->getCols() * a->getRows()) / double(threadsPerBlock * 2));
					if (blockNum <= 208) break;
				}

				YuiOkusora::Cuda::Matrix::__cudaMultiplyVal2Matrix << <blockNum, threadsPerBlock >> > (aPtr, bPtr, a->getRows(), a->getCols());

				hipDeviceSynchronize();

				hipMemcpy(a->getFlatted(), aPtr, sizeof(double) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
				hipFree(aPtr);
				hipFree(bPtr);
			}

			void multiplyMat2Matrix(Matrix* a, Matrix& b)
			{
				double* aPtr, * bPtr;
				hipMalloc(&aPtr, sizeof(double) * a->getCols() * a->getRows());
				hipMalloc(&bPtr, sizeof(double) * a->getCols() * a->getRows());

				hipMemcpy(aPtr, a->getFlatted(), sizeof(double) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
				hipMemcpy(bPtr, b.getFlatted(), sizeof(double) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);

				unsigned blockNum = 1, threadsPerBlock = 0;
				for (int i = 0; i <= 5; ++i) {
					threadsPerBlock = 32 << i;
					blockNum = ceil(double(a->getCols() * a->getRows()) / double(threadsPerBlock * 2));
					if (blockNum <= 208) break;
				}

				YuiOkusora::Cuda::Matrix::__cudaMultiplyMat2Matrix << <blockNum, threadsPerBlock >> > (aPtr, bPtr, a->getRows(), a->getCols());

				hipDeviceSynchronize();

				hipMemcpy(a->getFlatted(), aPtr, sizeof(double) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
				hipFree(aPtr);
				hipFree(bPtr);
			}

			void dotProductMatrix(Matrix* a, Matrix& b)
			{
				double* aPtr, * bPtr, * cPtr;

				hipMalloc(&aPtr, sizeof(double) * a->getRows() * a->getCols());
				hipMalloc(&bPtr, sizeof(double) * b.getRows() * b.getCols());
				hipMalloc(&cPtr, sizeof(double) * a->getRows() * b.getCols());

				hipMemset(cPtr, 0, sizeof(double) * a->getRows() * b.getCols());
				hipMemcpy(aPtr, a->getFlatted(), sizeof(double) * a->getRows() * a->getCols(), hipMemcpyHostToDevice);
				hipMemcpy(bPtr, b.getFlatted(), sizeof(double) * b.getRows() * b.getCols(), hipMemcpyHostToDevice);

				dim3 blockNum(0,0,1), threadNum(0,0,1);

				for (int i = 1; i <= 32; i = i << 1)
				{
					threadNum.x = i;
					blockNum.x = (unsigned)ceil(double(a->getRows()) / double(threadNum.x));
					if (blockNum.x <= 26) break;
				}

				for (int i = 1; i <= 32; i = i << 1)
				{
					threadNum.y = i;
					blockNum.y = (unsigned)ceil(double(b.getCols()) / double(threadNum.y));
					if (blockNum.y <= 26) break;
				}

				YuiOkusora::Cuda::Matrix::__cudaDotProductMatrix<<<blockNum, threadNum>>>(aPtr, bPtr, cPtr, a->getRows(), a->getCols(), b.getCols());

				free(a->getFlatted());

				a->getDataPtr() = (double*)malloc(sizeof(double) * a->getRows() * b.getCols());

				hipMemcpy(a->getFlatted(), cPtr, sizeof(double) * a->getRows() * b.getCols(), hipMemcpyDeviceToHost);

				hipFree(aPtr);
				hipFree(bPtr);
				hipFree(cPtr);
			}

			void rotateMatrix180(Matrix* a)
			{
				double* aPtr;
				hipMalloc(&aPtr, sizeof(double) * a->getCols() * a->getRows());
				
				hipMemcpy(aPtr, a->getFlatted(), sizeof(double) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);

				unsigned blockNum = 1, threadsPerBlock = 0;
				for (int i = 0; i <= 5; ++i) {
					threadsPerBlock = 32 << i;
					blockNum = ceil(double(a->getCols() * a->getRows()) / double(threadsPerBlock * 4));
					if (blockNum <= 208) break;
				}

				YuiOkusora::Cuda::Matrix::__cudaRotateMatrix180 << <blockNum, threadsPerBlock >> > (aPtr, a->getRows(), a->getCols());

				hipDeviceSynchronize();

				hipMemcpy(a->getFlatted(), aPtr, sizeof(double) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
				hipFree(aPtr);
			}
		}
	}
};