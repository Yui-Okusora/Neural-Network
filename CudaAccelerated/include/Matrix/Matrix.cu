#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Matrix.hpp"
#include <chrono>
#include <iostream>

#define TILE_WIDTH constexpr 32

namespace YuiOkusora
{
	namespace Cuda
	{
		namespace Matrix
		{
			__global__ static void __cudaAddMatrix(float* a, float *b, size_t rows, size_t cols)
			{
				unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;

				if (i < rows * cols)
					a[i] = a[i] + b[i];
				if (i < rows * cols - 1)
					a[i + 1] = a[i + 1] + b[i + 1];
			}

			__global__ static void __cudaAddVal2Matrix(float* a, float* b, size_t rows, size_t cols)
			{
				unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;

				if (i < rows * cols)
					a[i] = a[i] + *b;
				if (i < rows * cols - 1)
					a[i + 1] = a[i + 1] + *b;
			}

			__global__ static void __cudaSubtractMatrix(float* a, float* b, size_t rows, size_t cols)
			{
				unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;

				if (i < rows * cols)
					a[i] = a[i] - b[i];
				if (i < rows * cols - 1)
					a[i + 1] = a[i + 1] - b[i + 1];
			}
			
			__global__ static void __cudaTranspose(float* a, float* b, size_t rows, size_t cols)
			{
				unsigned i = (threadIdx.x + blockDim.x * blockIdx.x); // rows
				unsigned j = (threadIdx.y + blockDim.y * blockIdx.y); // cols

				if (i < rows && j < cols) {
					unsigned idx = i * cols + j;
					unsigned ridx = j * rows + i;

					b[ridx] = a[idx];
				}

			}

			__global__ static void __cudaMultiplyVal2Matrix(float* a, float* b, size_t rows, size_t cols)
			{
				unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;

				if (i < rows * cols)
					a[i] = a[i] * *b;
				if (i < rows * cols - 1)
					a[i + 1] = a[i + 1] * *b;
			}

			__global__ static void __cudaMultiplyMat2Matrix(float* a, float* b, size_t rows, size_t cols)
			{
				unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;

				if (i < rows * cols)
					a[i] = a[i] * b[i];
				if (i < rows * cols - 1)
					a[i + 1] = a[i + 1] * b[i + 1];
			}

			__global__ static void __cudaRotateMatrix180(float* a, size_t rows, size_t cols)
			{
				unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;
				float tmp = 0;
				const size_t size = rows * cols;
				if (i < size / 2)
				{
					tmp = a[i];
					a[i] = a[size - i - 1];
					a[size - i - 1] = tmp;
				}
					
				if (i < (size / 2) - 1)
				{
					tmp = a[i + 1];
					a[i + 1] = a[size - i - 2];
					a[size - i - 2] = tmp;
				}
			}

			__global__ static void __cudaDotProductMatrix(float* a, float* b, float* c, size_t Arows, size_t Acols, size_t Bcols)
			{
				unsigned i = threadIdx.y + blockIdx.y * blockDim.y; // rows
				unsigned j = threadIdx.x + blockIdx.x * blockDim.x; // cols
				unsigned k = threadIdx.z + blockIdx.z * blockDim.z;

				if (i >= Arows || j >= Bcols) return;

				c[i * Bcols + j] += a[i * Acols + k] * b[k * Bcols + j];
			}
			
		}
	}
	namespace Math
	{
		namespace Mat
		{
			void addMatrix(Matrix *a, Matrix &b)
			{
				float* aPtr, *bPtr;
				hipMalloc(&aPtr, sizeof(float) * a->getCols() * a->getRows());
				hipMalloc(&bPtr, sizeof(float) * a->getCols() * a->getRows());

				ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * a->getCols() * a->getRows());
				ViewOfAdvancedMemory& viewB = b.load(0, sizeof(float) * a->getCols() * a->getRows());

				hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
				hipMemcpy(bPtr, b.getViewPtr(viewB), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
				b.unload(viewB.lpMapAddress);

				unsigned blockNum = 1, threadsPerBlock = 0;
				for (int i = 0; i <= 5; ++i) {
					threadsPerBlock = 32 << i;
					blockNum = ceil(float(a->getCols() * a->getRows()) / float(threadsPerBlock * 2));
					if (blockNum <= 208) break;
				}

				YuiOkusora::Cuda::Matrix::__cudaAddMatrix<<<blockNum, threadsPerBlock>>>(aPtr, bPtr, a->getRows(), a->getCols());
				
				hipDeviceSynchronize();
				
				hipMemcpy(a->getViewPtr(viewA), aPtr, sizeof(float) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
				a->unload(viewA.lpMapAddress);
				
				hipFree(aPtr);
				hipFree(bPtr);
				
			}

			void addVal2Matrix(Matrix* a, const float &b)
			{
				float* aPtr, * bPtr;
				hipMalloc(&aPtr, sizeof(float) * a->getCols() * a->getRows());
				hipMalloc(&bPtr, sizeof(float));

				ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * a->getCols() * a->getRows());

				hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
				hipMemcpy(bPtr, &b, sizeof(float), hipMemcpyHostToDevice);

				unsigned blockNum = 1, threadsPerBlock = 0;
				for (int i = 0; i <= 5; ++i) {
					threadsPerBlock = 32 << i;
					blockNum = ceil(float(a->getCols() * a->getRows()) / float(threadsPerBlock * 2));
					if (blockNum <= 208) break;
				}

				YuiOkusora::Cuda::Matrix::__cudaAddVal2Matrix << <blockNum, threadsPerBlock >> > (aPtr, bPtr, a->getRows(), a->getCols());

				hipDeviceSynchronize();

				hipMemcpy(a->getViewPtr(viewA), aPtr, sizeof(float) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
				a->unload(viewA.lpMapAddress);
				
				hipFree(aPtr);
				hipFree(bPtr);

			}

			void subtractMatrix(Matrix* a, Matrix& b)
			{
				float* aPtr, * bPtr;
				hipMalloc(&aPtr, sizeof(float) * a->getCols() * a->getRows());
				hipMalloc(&bPtr, sizeof(float) * a->getCols() * a->getRows());

				ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * a->getCols() * a->getRows());
				ViewOfAdvancedMemory& viewB = b.load(0, sizeof(float) * a->getCols() * a->getRows());

				hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
				hipMemcpy(bPtr, b.getViewPtr(viewB), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
				b.unload(viewB.lpMapAddress);

				unsigned blockNum = 1, threadsPerBlock = 0;
				for (int i = 0; i <= 5; ++i) {
					threadsPerBlock = 32 << i;
					blockNum = ceil(float(a->getCols() * a->getRows()) / float(threadsPerBlock * 2));
					if (blockNum <= 208) break;
				}

				YuiOkusora::Cuda::Matrix::__cudaSubtractMatrix << <blockNum, threadsPerBlock >> > (aPtr, bPtr, a->getRows(), a->getCols());

				hipDeviceSynchronize();

				hipMemcpy(a->getViewPtr(viewA), aPtr, sizeof(float) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
				a->unload(viewA.lpMapAddress);
				
				hipFree(aPtr);
				hipFree(bPtr);

			}
		
			void transposeMatrix(Matrix* a)
			{
				float* aPtr, *bPtr;
				hipMalloc(&aPtr, sizeof(float) * a->getCols() * a->getRows());
				hipMalloc(&bPtr, sizeof(float) * a->getCols() * a->getRows());

				ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * a->getCols() * a->getRows());

				hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);

				dim3 blockNum(0,0,1), threadNum(0,0,1);
				for (int i = 1; i <= 32;++i)
				{
					threadNum.x = i;
					blockNum.x = ceil(float(a->getRows()) / float(i));
					if (blockNum.x <= 208) break;
				}
				
				for (int i = 1; i <= 32;++i)
				{
					threadNum.y = i;
					blockNum.y = ceil(float(a->getCols()) / float(i));
					if (blockNum.y <= 208) break;
				}

				YuiOkusora::Cuda::Matrix::__cudaTranspose<<<blockNum, threadNum>>>(aPtr, bPtr, a->getRows(), a->getCols());

				hipDeviceSynchronize();

				hipMemcpy(a->getViewPtr(viewA), bPtr, sizeof(float) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
				a->unload(viewA.lpMapAddress);

				hipFree(aPtr);
				hipFree(bPtr);

			}
		
			void multiplyVal2Matrix(Matrix* a, const float& b)
			{
				float* aPtr, * bPtr;
				hipMalloc(&aPtr, sizeof(float) * a->getCols() * a->getRows());
				hipMalloc(&bPtr, sizeof(float));

				ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * a->getCols() * a->getRows());

				hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
				hipMemcpy(bPtr, &b, sizeof(float), hipMemcpyHostToDevice);

				unsigned blockNum = 1, threadsPerBlock = 0;
				for (int i = 0; i <= 5; ++i) {
					threadsPerBlock = 32 << i;
					blockNum = ceil(float(a->getCols() * a->getRows()) / float(threadsPerBlock * 2));
					if (blockNum <= 208) break;
				}

				YuiOkusora::Cuda::Matrix::__cudaMultiplyVal2Matrix << <blockNum, threadsPerBlock >> > (aPtr, bPtr, a->getRows(), a->getCols());

				hipDeviceSynchronize();

				hipMemcpy(a->getViewPtr(viewA), aPtr, sizeof(float) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
				a->unload(viewA.lpMapAddress);
				
				hipFree(aPtr);
				hipFree(bPtr);
			}

			void multiplyMat2Matrix(Matrix* a, Matrix& b)
			{
				float* aPtr, * bPtr;
				hipMalloc(&aPtr, sizeof(float) * a->getCols() * a->getRows());
				hipMalloc(&bPtr, sizeof(float) * b.getCols() * b.getRows());

				ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * a->getCols() * a->getRows());
				ViewOfAdvancedMemory& viewB = b.load(0, sizeof(float) * b.getCols() * b.getRows());

				hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
				hipMemcpy(bPtr, b.getViewPtr(viewB), sizeof(float) * b.getCols() * b.getRows(), hipMemcpyHostToDevice);

				b.unload(viewB.lpMapAddress);

				unsigned blockNum = 1, threadsPerBlock = 0;
				for (int i = 0; i <= 5; ++i) {
					threadsPerBlock = 32 << i;
					blockNum = ceil(float(a->getCols() * a->getRows()) / float(threadsPerBlock * 2));
					if (blockNum <= 208) break;
				}

				YuiOkusora::Cuda::Matrix::__cudaMultiplyMat2Matrix << <blockNum, threadsPerBlock >> > (aPtr, bPtr, a->getRows(), a->getCols());

				hipDeviceSynchronize();

				hipMemcpy(a->getViewPtr(viewA), aPtr, sizeof(float) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
				a->unload(viewA.lpMapAddress);
				
				hipFree(aPtr);
				hipFree(bPtr);
			}

			void dotProductMatrix(Matrix* a, Matrix& b)
			{
				float* aPtr = nullptr, * bPtr = nullptr, * cPtr = nullptr;
				auto aSIZE = a->getRows() * a->getCols(), bSIZE = b.getRows() * b.getCols(), cSIZE = a->getRows() * b.getCols();

				std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();

				hipMalloc(&aPtr, sizeof(float) * (static_cast<size_t>(aSIZE) + bSIZE + cSIZE));
				bPtr = aPtr + bSIZE;
				cPtr = aPtr + cSIZE;

				hipMemset(cPtr, 0, sizeof(float) * cSIZE);

				ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * aSIZE);
				ViewOfAdvancedMemory& viewB = b.load(0, sizeof(float) * bSIZE);

				hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * aSIZE, hipMemcpyHostToDevice);
				hipMemcpy(bPtr, b.getViewPtr(viewB), sizeof(float) * bSIZE, hipMemcpyHostToDevice);

				a->unload(viewA.lpMapAddress);
				b.unload(viewB.lpMapAddress);

				std::chrono::high_resolution_clock::time_point end = std::chrono::high_resolution_clock::now();

				std::cout << std::chrono::duration<long long, std::nano>(end - start).count() << "\n";

				dim3 blockNum(0,0,1), threadNum(0,0,1);

				for (unsigned i = 1; i <= 16; i = i << 2)
				{
					threadNum.z = i;
					blockNum.z = (unsigned)ceil(float(a->getCols()) / float(threadNum.z));
					if (blockNum.z <= 26) break;
				}

				for (unsigned i = 1; i <= 32 / sqrt(threadNum.z); i = i << 1)
				{
					threadNum.y = i;
					blockNum.y = (unsigned)ceil(float(a->getRows()) / float(threadNum.y));
					if (blockNum.y <= 26) break;
				}

				for (unsigned i = 1; i <= 1024; i = i << 1)
				{
					float tmp = i / float(threadNum.y * threadNum.z);
					blockNum.x = (unsigned)ceil(float(b.getCols()) / tmp);
					threadNum.x = (unsigned)ceil(tmp);
					if (blockNum.x <= 26) break;
				}

				//threadNum.y = 1024 / threadNum.x;
				//blockNum.y = (unsigned)ceil(float(b.getCols()) / float(threadNum.y));

				YuiOkusora::Cuda::Matrix::__cudaDotProductMatrix<<<blockNum, threadNum>>>(aPtr, bPtr, cPtr, a->getRows(), a->getCols(), b.getCols());

				hipDeviceSynchronize();

				a->resize(cSIZE);
				ViewOfAdvancedMemory& viewC = a->load(0, sizeof(float) * cSIZE);

				hipMemcpy(a->getViewPtr(viewC), cPtr, sizeof(float) * cSIZE, hipMemcpyDeviceToHost);
				a->unload(viewC.lpMapAddress);

				hipFree(aPtr);
				hipFree(bPtr);
				hipFree(cPtr);
			}

			void rotateMatrix180(Matrix* a)
			{
				float* aPtr;
				hipMalloc(&aPtr, sizeof(float) * a->getCols() * a->getRows());

				ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * a->getCols() * a->getRows());
				
				hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);

				unsigned blockNum = 1, threadsPerBlock = 0;
				for (int i = 0; i <= 5; ++i) {
					threadsPerBlock = 32 << i;
					blockNum = ceil(float(a->getCols() * a->getRows()) / float(threadsPerBlock * 4));
					if (blockNum <= 208) break;
				}

				YuiOkusora::Cuda::Matrix::__cudaRotateMatrix180 << <blockNum, threadsPerBlock >> > (aPtr, a->getRows(), a->getCols());

				hipDeviceSynchronize();

				hipMemcpy(a->getViewPtr(viewA), aPtr, sizeof(float) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
				a->unload(viewA.lpMapAddress);
				
				hipFree(aPtr);
			}
		}
	}
};