﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include "iostream"

void loadMem()
{
	float* ptr;
	hipMalloc(&ptr, sizeof(float));
	hipFree(ptr);
}

void resetGPU()
{
	/*cudaDeviceProp DeviceProperties;
	cudaGetDeviceProperties(&DeviceProperties, 0);
	int GPUcount = 0;
	cudaGetDeviceCount(&GPUcount);
	std::cout << GPUcount;*/
	hipDeviceReset();
}