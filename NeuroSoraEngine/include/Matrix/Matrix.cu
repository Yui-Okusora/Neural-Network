#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Matrix.hpp"

namespace NeuroSoraCore
{

	namespace Internal
	{
		namespace Cuda
		{
			namespace Math
			{
				__host__ __device__ float activationFunction(float x, const ActivationType activationType, float reluParam)
				{
					switch (activationType)
					{
					case ActivationType::Tanh:
						return tanhf(x);
						break;
					case ActivationType::Sigmoid:
						return 1.0f / (1.0f + expf(-x));
						break;
					case ActivationType::ReLU:
						return max(0.0f, x);
						break;
					case ActivationType::LReLU:
						return (x > 0.0f) ? x : reluParam * x;
						break;
					default:
						return 0.0f;
						break;
					}
				}

				__host__ __device__ float activationFunctionDerivative(float x, const ActivationType activationType, float reluParam)
				{
					switch (activationType)
					{
					case ActivationType::Tanh:
						return 1.0f - x * x;
						break;
					case ActivationType::Sigmoid:
						return x * (1.0f - x);
						break;
					case ActivationType::ReLU:
						return (x > 0.0f) ? 1.0f : 0.0f;
						break;
					case ActivationType::LReLU:
						return (x > 0.0f) ? 1.0f : reluParam;
					default:
						return 0.0f;
						break;
					}
				}
			}
			namespace Matrix
			{
				__global__ static void __cudaAddMatrix(float* a, float* b, size_t rows, size_t cols)
				{
					unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;

					if (i < rows * cols)
						a[i] = a[i] + b[i];
					if (i < rows * cols - 1)
						a[i + 1] = a[i + 1] + b[i + 1];
				}

				__global__ static void __cudaAddVal2Matrix(float* a, float* b, size_t rows, size_t cols)
				{
					unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;

					if (i < rows * cols)
						a[i] = a[i] + *b;
					if (i < rows * cols - 1)
						a[i + 1] = a[i + 1] + *b;
				}

				__global__ static void __cudaSubtractMatrix(float* a, float* b, size_t rows, size_t cols)
				{
					unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;

					if (i < rows * cols)
						a[i] = a[i] - b[i];
					if (i < rows * cols - 1)
						a[i + 1] = a[i + 1] - b[i + 1];
				}

				__global__ static void __cudaTranspose(float* a, float* b, size_t rows, size_t cols)
				{
					unsigned i = (threadIdx.x + blockDim.x * blockIdx.x); // rows
					unsigned j = (threadIdx.y + blockDim.y * blockIdx.y); // cols

					if (i < rows && j < cols) {
						unsigned idx = i * cols + j;
						unsigned ridx = j * rows + i;

						b[ridx] = a[idx];
					}

				}

				__global__ static void __cudaMultiplyVal2Matrix(float* a, float* b, size_t rows, size_t cols)
				{
					unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;

					if (i < rows * cols)
						a[i] = a[i] * *b;
					if (i < rows * cols - 1)
						a[i + 1] = a[i + 1] * *b;
				}

				__global__ static void __cudaMultiplyMat2Matrix(float* a, float* b, size_t rows, size_t cols)
				{
					unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;

					if (i < rows * cols)
						a[i] = a[i] * b[i];
					if (i < rows * cols - 1)
						a[i + 1] = a[i + 1] * b[i + 1];
				}

				__global__ static void __cudaRotateMatrix180(float* a, size_t rows, size_t cols)
				{
					unsigned i = (threadIdx.x + blockDim.x * blockIdx.x) * 2;
					float tmp = 0;
					const size_t size = rows * cols;
					if (i < size / 2)
					{
						tmp = a[i];
						a[i] = a[size - i - 1];
						a[size - i - 1] = tmp;
					}

					if (i < (size / 2) - 1)
					{
						tmp = a[i + 1];
						a[i + 1] = a[size - i - 2];
						a[size - i - 2] = tmp;
					}
				}

				__global__ static void __cudaDotProductMatrix(float* a, float* b, float* c, size_t Arows, size_t Acols, size_t Bcols)
				{
					unsigned i = threadIdx.y + blockIdx.y * blockDim.y; // rows 4
					unsigned j = threadIdx.x + blockIdx.x * blockDim.x; // cols 3
					unsigned k = threadIdx.z + blockIdx.z * blockDim.z; //      3

					if (i >= Arows || j >= Bcols || k >= Acols) return;

					atomicAdd(&c[i * Bcols + j], a[i * Acols + k] * b[k * Bcols + j]);
				}

				__global__ static void __cudaApplyActivation(float* a, size_t rows, size_t cols, ActivationType acti, float reluParam = 0.15f)
				{
					unsigned i = threadIdx.x + blockDim.x * blockIdx.x;

					if (i >= rows * cols) return;

					a[i] = Internal::Cuda::Math::activationFunction(a[i], acti, reluParam);
				}

				__global__ static void __cudaApplyActivationDerivative(float* a, float* scalar, size_t rows, size_t cols, ActivationType acti, float reluParam = 0.15f)
				{
					unsigned i = threadIdx.x + blockDim.x * blockIdx.x;

					if (i >= rows * cols) return;

					if (scalar == NULL)
						a[i] = Internal::Cuda::Math::activationFunctionDerivative(a[i], acti, reluParam);
					else
						a[i] = scalar[i] * Internal::Cuda::Math::activationFunctionDerivative(a[i], acti, reluParam);
				}
			}
		}
		namespace Math
		{
			namespace Mat
			{
				void addMatrix(Matrix* a, Matrix& b)
				{
					float* aPtr, * bPtr;
					hipMalloc(&aPtr, sizeof(float) * a->getCols() * a->getRows());
					hipMalloc(&bPtr, sizeof(float) * a->getCols() * a->getRows());

					ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * a->getCols() * a->getRows());
					ViewOfAdvancedMemory& viewB = b.load(0, sizeof(float) * a->getCols() * a->getRows());

					hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
					hipMemcpy(bPtr, b.getViewPtr(viewB), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
					b.unload(viewB);

					unsigned blockNum = 1, threadsPerBlock = 0;
					for (int i = 0; i <= 5; ++i) {
						threadsPerBlock = 32 << i;
						blockNum = static_cast<unsigned>(ceil(float(a->getCols() * a->getRows()) / float(threadsPerBlock * 2)));
						if (blockNum <= 208) break;
					}

					Internal::Cuda::Matrix::__cudaAddMatrix << <blockNum, threadsPerBlock >> > (aPtr, bPtr, a->getRows(), a->getCols());

					hipDeviceSynchronize();

					hipMemcpy(a->getViewPtr(viewA), aPtr, sizeof(float) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
					a->unload(viewA);

					hipFree(aPtr);
					hipFree(bPtr);

				}

				void addVal2Matrix(Matrix* a, const float& b)
				{
					float* aPtr, * bPtr;
					hipMalloc(&aPtr, sizeof(float) * a->getCols() * a->getRows());
					hipMalloc(&bPtr, sizeof(float));

					ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * a->getCols() * a->getRows());

					hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
					hipMemcpy(bPtr, &b, sizeof(float), hipMemcpyHostToDevice);

					unsigned blockNum = 1, threadsPerBlock = 0;
					for (int i = 0; i <= 5; ++i) {
						threadsPerBlock = 32 << i;
						blockNum = static_cast<unsigned>(ceil(float(a->getCols() * a->getRows()) / float(threadsPerBlock * 2)));
						if (blockNum <= 208) break;
					}

					Internal::Cuda::Matrix::__cudaAddVal2Matrix << <blockNum, threadsPerBlock >> > (aPtr, bPtr, a->getRows(), a->getCols());

					hipDeviceSynchronize();

					hipMemcpy(a->getViewPtr(viewA), aPtr, sizeof(float) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
					a->unload(viewA);

					hipFree(aPtr);
					hipFree(bPtr);

				}

				void subtractMatrix(Matrix* a, Matrix& b)
				{
					float* aPtr, * bPtr;
					hipMalloc(&aPtr, sizeof(float) * a->getCols() * a->getRows());
					hipMalloc(&bPtr, sizeof(float) * a->getCols() * a->getRows());

					ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * a->getCols() * a->getRows());
					ViewOfAdvancedMemory& viewB = b.load(0, sizeof(float) * a->getCols() * a->getRows());

					hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
					hipMemcpy(bPtr, b.getViewPtr(viewB), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
					b.unload(viewB);

					unsigned blockNum = 1, threadsPerBlock = 0;
					for (int i = 0; i <= 5; ++i) {
						threadsPerBlock = 32 << i;
						blockNum = static_cast<unsigned>(ceil(float(a->getCols() * a->getRows()) / float(threadsPerBlock * 2)));
						if (blockNum <= 208) break;
					}

					Internal::Cuda::Matrix::__cudaSubtractMatrix << <blockNum, threadsPerBlock >> > (aPtr, bPtr, a->getRows(), a->getCols());

					hipDeviceSynchronize();

					hipMemcpy(a->getViewPtr(viewA), aPtr, sizeof(float) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
					a->unload(viewA);

					hipFree(aPtr);
					hipFree(bPtr);

				}

				void transposeMatrix(Matrix* a)
				{
					float* aPtr, * bPtr;
					hipMalloc(&aPtr, sizeof(float) * a->getCols() * a->getRows());
					hipMalloc(&bPtr, sizeof(float) * a->getCols() * a->getRows());

					ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * a->getCols() * a->getRows());

					hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);

					dim3 blockNum(0, 0, 1), threadNum(0, 0, 1);
					for (int i = 1; i <= 32; ++i)
					{
						threadNum.x = i;
						blockNum.x = static_cast<unsigned>(ceil(float(a->getRows()) / float(i)));
						if (blockNum.x <= 208) break;
					}

					for (int i = 1; i <= 32; ++i)
					{
						threadNum.y = i;
						blockNum.y = static_cast<unsigned>(ceil(float(a->getCols()) / float(i)));
						if (blockNum.y <= 208) break;
					}

					Internal::Cuda::Matrix::__cudaTranspose << <blockNum, threadNum >> > (aPtr, bPtr, a->getRows(), a->getCols());

					hipDeviceSynchronize();

					hipMemcpy(a->getViewPtr(viewA), bPtr, sizeof(float) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
					a->unload(viewA);

					hipFree(aPtr);
					hipFree(bPtr);

				}

				void multiplyVal2Matrix(Matrix* a, const float& b)
				{
					float* aPtr, * bPtr;
					hipMalloc(&aPtr, sizeof(float) * a->getCols() * a->getRows());
					hipMalloc(&bPtr, sizeof(float));

					ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * a->getCols() * a->getRows());

					hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
					hipMemcpy(bPtr, &b, sizeof(float), hipMemcpyHostToDevice);

					unsigned blockNum = 1, threadsPerBlock = 0;
					for (int i = 0; i <= 5; ++i) {
						threadsPerBlock = 32 << i;
						blockNum = static_cast<unsigned>(ceil(float(a->getCols() * a->getRows()) / float(threadsPerBlock * 2)));
						if (blockNum <= 208) break;
					}

					Internal::Cuda::Matrix::__cudaMultiplyVal2Matrix << <blockNum, threadsPerBlock >> > (aPtr, bPtr, a->getRows(), a->getCols());

					hipDeviceSynchronize();

					hipMemcpy(a->getViewPtr(viewA), aPtr, sizeof(float) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
					a->unload(viewA);

					hipFree(aPtr);
					hipFree(bPtr);
				}

				void multiplyMat2Matrix(Matrix* a, Matrix& b)
				{
					float* aPtr, * bPtr;
					hipMalloc(&aPtr, sizeof(float) * a->getCols() * a->getRows());
					hipMalloc(&bPtr, sizeof(float) * b.getCols() * b.getRows());

					ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * a->getCols() * a->getRows());
					ViewOfAdvancedMemory& viewB = b.load(0, sizeof(float) * b.getCols() * b.getRows());

					hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);
					hipMemcpy(bPtr, b.getViewPtr(viewB), sizeof(float) * b.getCols() * b.getRows(), hipMemcpyHostToDevice);

					b.unload(viewB);

					unsigned blockNum = 1, threadsPerBlock = 0;
					for (int i = 0; i <= 5; ++i) {
						threadsPerBlock = 32 << i;
						blockNum = static_cast<unsigned>(ceil(float(a->getCols() * a->getRows()) / float(threadsPerBlock * 2)));
						if (blockNum <= 208) break;
					}

					Internal::Cuda::Matrix::__cudaMultiplyMat2Matrix << <blockNum, threadsPerBlock >> > (aPtr, bPtr, a->getRows(), a->getCols());

					hipDeviceSynchronize();

					hipMemcpy(a->getViewPtr(viewA), aPtr, sizeof(float) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
					a->unload(viewA);

					hipFree(aPtr);
					hipFree(bPtr);
				}

				void dotProductMatrix(Matrix* a, const Matrix& b)
				{
					float* aPtr = nullptr, * bPtr = nullptr, * cPtr = nullptr;
					auto aSIZE = a->getRows() * a->getCols();
					auto bSIZE = b.getRows() * b.getCols();
					auto cSIZE = a->getRows() * b.getCols();

					hipMalloc(&aPtr, sizeof(float) * aSIZE);
					hipMalloc(&bPtr, sizeof(float) * bSIZE);
					hipMalloc(&cPtr, sizeof(float) * cSIZE);

					hipMemset(cPtr, 0, sizeof(float) * cSIZE);

					ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * aSIZE);
					ViewOfAdvancedMemory& viewB = const_cast<Matrix&>(b).load(0, sizeof(float) * bSIZE);

					hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * aSIZE, hipMemcpyHostToDevice);
					hipMemcpy(bPtr, const_cast<Matrix&>(b).getViewPtr(viewB), sizeof(float) * bSIZE, hipMemcpyHostToDevice);

					a->unload(viewA);
					const_cast<Matrix&>(b).unload(viewB);

					dim3 blockNum(1, 1, 1), threadNum(1, 1, 1);

					for (unsigned i = 1; i <= 16; i = i << 2)
					{
						threadNum.z = i;
						blockNum.z = (unsigned)ceil(float(a->getCols()) / float(threadNum.z));
						if (blockNum.z <= 26) break;
					}

					for (unsigned i = 1; i <= 32 / sqrt(threadNum.z); i = i << 1)
					{
						threadNum.y = i;
						blockNum.y = (unsigned)ceil(float(a->getRows()) / float(threadNum.y));
						if (blockNum.y <= 26) break;
					}

					for (unsigned i = 1; i <= 1024; i = i << 1)
					{
						float tmp = i / float(threadNum.y * threadNum.z);
						blockNum.x = (unsigned)ceil(float(b.getCols()) / tmp);
						threadNum.x = (unsigned)ceil(tmp);
						if (blockNum.x <= 26) break;
					}

					Internal::Cuda::Matrix::__cudaDotProductMatrix << <blockNum, threadNum >> > (aPtr, bPtr, cPtr, a->getRows(), a->getCols(), b.getCols());

					a->resize(cSIZE);
					ViewOfAdvancedMemory& viewC = a->load(0, sizeof(float) * cSIZE);

					hipDeviceSynchronize();

					hipMemcpy(a->getViewPtr(viewC), cPtr, sizeof(float) * cSIZE, hipMemcpyDeviceToHost);
					a->unload(viewC);

					hipFree(aPtr);
					hipFree(bPtr);
					hipFree(cPtr);
				}

				void rotateMatrix180(Matrix* a)
				{
					float* aPtr;
					hipMalloc(&aPtr, sizeof(float) * a->getCols() * a->getRows());

					ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * a->getCols() * a->getRows());

					hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);

					unsigned blockNum = 1, threadsPerBlock = 0;
					for (int i = 0; i <= 5; ++i) {
						threadsPerBlock = 32 << i;
						blockNum = static_cast<unsigned>(ceil(float(a->getCols() * a->getRows()) / float(threadsPerBlock * 4)));
						if (blockNum <= 208) break;
					}

					Internal::Cuda::Matrix::__cudaRotateMatrix180 << <blockNum, threadsPerBlock >> > (aPtr, a->getRows(), a->getCols());

					hipDeviceSynchronize();

					hipMemcpy(a->getViewPtr(viewA), aPtr, sizeof(float) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
					a->unload(viewA);

					hipFree(aPtr);
				}

				void applyActivationMatrix(Matrix* a, ActivationType acti, float reluParam)
				{
					float* aPtr;
					hipMalloc(&aPtr, sizeof(float) * a->getCols() * a->getRows());

					ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * a->getCols() * a->getRows());

					hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * a->getCols() * a->getRows(), hipMemcpyHostToDevice);

					unsigned blockNum = 1, threadsPerBlock = 0;
					for (int i = 0; i <= 5; ++i) {
						threadsPerBlock = 32 << i;
						blockNum = static_cast<unsigned>(ceil(float(a->getCols() * a->getRows()) / float(threadsPerBlock)));
						if (blockNum <= 208) break;
					}

					Internal::Cuda::Matrix::__cudaApplyActivation << <blockNum, threadsPerBlock >> > (aPtr, a->getRows(), a->getCols(), acti, reluParam);

					hipDeviceSynchronize();

					hipMemcpy(a->getViewPtr(viewA), aPtr, sizeof(float) * a->getCols() * a->getRows(), hipMemcpyDeviceToHost);
					a->unload(viewA);

					hipFree(aPtr);
				}

				void applyActivationDerivativeMatrix(Matrix* a, ActivationType acti, float reluParam, Matrix* scalar)
				{
					float* aPtr = NULL, * scalarPtr = NULL;
					size_t aSIZE = a->getCols() * a->getRows();

					hipMalloc(&aPtr, sizeof(float) * aSIZE);

					if (scalar != NULL)
					{
						hipMalloc(&scalarPtr, sizeof(float) * aSIZE); // a dim = scalar dim
						ViewOfAdvancedMemory& viewScalar = scalar->load(0, sizeof(float) * aSIZE);

						hipMemcpy(scalarPtr, scalar->getViewPtr(viewScalar), sizeof(float) * aSIZE, hipMemcpyHostToDevice);

						scalar->unload(viewScalar);
					}

					ViewOfAdvancedMemory& viewA = a->load(0, sizeof(float) * aSIZE);

					hipMemcpy(aPtr, a->getViewPtr(viewA), sizeof(float) * aSIZE, hipMemcpyHostToDevice);

					unsigned blockNum = 1, threadsPerBlock = 0;
					for (int i = 0; i <= 5; ++i) {
						threadsPerBlock = 32 << i;
						blockNum = static_cast<unsigned>(ceil(float(aSIZE) / float(threadsPerBlock)));
						if (blockNum <= 208) break;
					}

					Internal::Cuda::Matrix::__cudaApplyActivationDerivative << <blockNum, threadsPerBlock >> > (aPtr, scalarPtr, a->getRows(), a->getCols(), acti, reluParam);

					hipDeviceSynchronize();

					hipMemcpy(a->getViewPtr(viewA), aPtr, sizeof(float) * aSIZE, hipMemcpyDeviceToHost);
					a->unload(viewA);

					hipFree(aPtr);
					if (scalarPtr != NULL)
						hipFree(scalarPtr);
				}

			}
		}
	}
}